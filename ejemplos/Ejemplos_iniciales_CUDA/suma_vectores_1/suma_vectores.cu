
#include <hip/hip_runtime.h>
#include "stdio.h"

const int N=64;

__global__ void VecAdd( float *A, float *B, float *C, int Ntot)
{
int i=threadIdx.x;
C[i]=A[i]+B[i];
}

int main()
{
/* pointers to host memory */
float *a, *b, *c;
/* pointers to device memory */
float *a_d, *b_d, *c_d;
int i;

/* Allocate arrays a, b and c on host*/
a = (float*) malloc(N*sizeof(float));
b = (float*) malloc(N*sizeof(float));
c = (float*) malloc(N*sizeof(float));

/* Allocate arrays a_d, b_d and c_d on device*/
hipMalloc ((void **) &a_d, sizeof(float)*N);
hipMalloc ((void **) &b_d, sizeof(float)*N);
hipMalloc ((void **) &c_d, sizeof(float)*N);

/* Initialize arrays a and b */
for (i=0; i<N;i++)
{
a[i]= (float) i;
b[i]= -(float) i;
}


/* Copy data from host memory to device memory */
hipMemcpy(a_d, a, sizeof(float)*N, hipMemcpyHostToDevice);
hipMemcpy(b_d, b, sizeof(float)*N, hipMemcpyHostToDevice);

/* Add arrays a and b, store result in c */
VecAdd<<< 1, N >>>(a_d, b_d, c_d, N);

/* Copy data from deveice memory to host memory */
hipMemcpy(c, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

/* Print c */
for (i=0; i<N;i++)
printf(" c[%d]=%f\n",i,c[i]);

/* Free the memory */
free(a); free(b); free(c);
hipFree(a_d); hipFree(b_d);hipFree(c_d);

}
